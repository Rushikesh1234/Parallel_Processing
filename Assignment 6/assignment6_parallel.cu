#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<math.h>

#define BLOCK_SIZE 16

float **dist;
int n, w;
float p;

__global__ void calculateDistance(float *dev_dist, size_t pitch, int n)
{
	// Initialize blocksize for each threadIdx to access row value in matrix
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	float *row;
	float a,b;
	float temp;
	
	// Now, we are accessing matrix row value, if i and j is less than row size of matrix
	if(i<n && j<n)
	{
		// Iterating through each array element
		for(int k=0; k<n;k++)
		{
			// Fetch a[k][j] value from our column of matrix
			row = (float*)((char*)dev_dist + k*pitch);
			a = row[j];

			// Fetch a[i][k] value from row of matrix for kth element
			row = (float*)((char*)dev_dist + i*pitch);
			b = row[k];
			
			// If our temp value is less than current postion row[j] (i.e. a[i][j]), then, we store caluclated value or minimum distance value
			temp = a + b;
			if(row[j] > temp)
			{
				row[j] = temp;
			}
		}
	}
}

int main()
{
	n = 8;
	w = 4;
	p = 2;
	
	// Declare array
	dist = (float **) malloc(n * sizeof(float*));
	
	for(int i=0; i<n; i++)
	{
		dist[i] = (float *) malloc(n * sizeof(float));
	}
	
	float rand;
	
	// Generate Matrix Data using Random Function
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			rand = ((float)rand()) / n;
			if(rand != 0)
			{
				dist[i][j] = rand;
			}
		}
		// Diagonal element is zero
		dist[i][i] = 0;
	}
	
	float *dev_dist;
	size_t size;
	
	// Allocate CUDA memory for datatypes
	hipMallocPitch(&dev_dist, &size, n*sizeof(float), n);
	
	// Copy our input data to CUDA variable
	hipMemcpy2D(dev_dist, size, dist, n*sizeof(float), n*sizeof(float), n, hipMemcpyHostToDevice);

	// Fetch threads and blocks for GPU  execution
	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 numBlocks(n / threadsPerBlock.x, n / threadsPerBlock.y);
	
	// Call calculateDistance function which is declared and executed in GPU
	calculateDistance<<<numBlocks, threadsPerBlock>>>(dev_dist, pitch, n);

	// Finally, cpy our output in final variable
	hipMemcpy2D(dist, n*sizeof(float), dev_dist, pitch, n*sizeof(float), n, hipMemcpyHostToDevice);
	
	// Free Datatype
	hipFree(dev_dist);
	
	return 0;
}
